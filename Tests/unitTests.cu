/**
 * Multiple tests to verify our implementation of the Dynamic Analysis tool. We hand
 * compute the expected number of global memory uncoalesce acceses.

 */

#include <hip/hip_runtime.h>
#include <stdio.h>
/* Number big enough to assure no out of bounds accesses. */
#define N 10000

void charTests();
void intTests();
void doubleTests();
void structTests();


__global__ void charAddOne(char* array, int stride);
__global__ void intAddOne(int* array, int stride);
__global__ void intAddOneHalf(int* array, int stride);
__global__ void intAddOneOff(int* array, int stride);
__global__ void intAddOneEvens(int* array, int stride);
__global__ void intAddOneOdds(int* array, int stride);
__global__ void intAddOneDiff(int* array, int stride);
__global__ void intAddOneSame(int* array, int stride);
__global__ void doubleAddOne(double* array, int stride);
__global__ void structAddOneX(struct myStruct* array, int stride);
__global__ void structAddOneY(struct myStruct* array, int stride);
__global__ void structAddOneZ(struct myStruct* array, int stride);

struct myStruct{
  int x;
  int y;
  int z;
};

char* printStr = "[Tests %d] Cache lines expected: %d\n\n";
char* printStrWarp = "[Tests %d] Warps printing expected: %d\n\n";

int main(){
  // Uncomment to try out!
  //  charTests();
  //intTests();
  //doubleTests();
  structTests();
  return 0;
}

/**
 * Tests for struct
 */
void structTests(){
  struct myStruct x[N] = {0};
  struct myStruct * xDev;
  hipMalloc(&xDev, sizeof(struct myStruct) * N);
  hipMemcpy(xDev, x, sizeof(struct myStruct)* N, hipMemcpyHostToDevice);
  printf("[Stuct Tests]\n\n");
  // Run multiple tests to ensure our pass is working!
  // Synchonize necessary so that CPU waits for kernel to finish before printing.
  
  // Test.
  { int testNum = 1;    int blocks = 1;
    int threads = 10;   int stride = 1;
    int cacheLines = 1;
    structAddOneX<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // Test.
  { int testNum = 2;    int blocks = 1;
    int threads = 11;   int stride = 1;
    int cacheLines = 1;
    structAddOneX<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // Test.
  { int testNum = 3;    int blocks = 1;
    int threads = 10;   int stride = 1;
    int cacheLines = 1;
    structAddOneY<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // Test.
  { int testNum = 4;    int blocks = 1;
    int threads = 11;   int stride = 1;
    int cacheLines = 1;
    structAddOneY<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // Test.
  { int testNum = 5;    int blocks = 1;
    int threads = 10;   int stride = 1;
    int cacheLines = 1;
    structAddOneZ<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // Test.
  { int testNum = 6;    int blocks = 1;
    int threads = 11;   int stride = 1;
    int cacheLines = 2;
    structAddOneZ<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }


  hipMemcpy(x, xDev, sizeof(struct myStruct) * N, hipMemcpyDeviceToHost);
  hipFree(x);
}


/**
 * Tests for doubles.
 */
void doubleTests(){
  double x[N] = {0};
  double * xDev;
  hipMalloc(&xDev, sizeof(double) * N);
  hipMemcpy(xDev, x, sizeof(double)* N, hipMemcpyHostToDevice);

  printf("[Double Tests]\n\n");
  // Run multiple tests to ensure our pass is working!
  // Synchonize necessary so that CPU waits for kernel to finish before printing.
  
  // Test.
  { int testNum = 1;    int blocks = 1;
    int threads = 32;   int stride = 1;
    int cacheLines = 2;
    doubleAddOne<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // Test.
  { int testNum = 1;    int blocks = 1;
    int threads = 16;   int stride = 1;
    int cacheLines = 1;
    doubleAddOne<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // Test.
  { int testNum = 1;    int blocks = 1;
    int threads = 17;   int stride = 1;
    int cacheLines = 2;
    doubleAddOne<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // Test.
  { int testNum = 1;    int blocks = 1;
    int threads = 32;   int stride = 2;
    int cacheLines = 4;
    doubleAddOne<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // Test.
  { int testNum = 1;    int blocks = 1;
    int threads = 8;   int stride = 2;
    int cacheLines = 1;
    doubleAddOne<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  hipMemcpy(x, xDev, sizeof(double) * N, hipMemcpyDeviceToHost);
  hipFree(x);
}

/**
 * Tests for accessing integers.
 */
void intTests(){
  int x[N] = {0};
  int* xDev;
  hipMalloc(&xDev, sizeof(int) * N);
  hipMemcpy(xDev, x, sizeof(int)* N, hipMemcpyHostToDevice);

  printf("[Int Tests]\n\n");
  // Run multiple tests to ensure our pass is working!
  // Synchonize necessary so that CPU waits for kernel to finish before printing.
  
  // Test. Needs 32 bytes of continuous memory.
  { int testNum = 1;    int blocks = 1;
    int threads = 32;   int stride = 1;
    int cacheLines = 1;
    intAddOne<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // Test.
  { int testNum = 2;    int blocks = 1;
    int threads = 32;   int stride = 2;
    int cacheLines = 2;
    intAddOne<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // Test.
  { int testNum = 3;    int blocks = 1;
    int threads = 16;   int stride = 2;
    int cacheLines = 1;
    intAddOne<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // Test.
  { int testNum = 4;    int blocks = 32;
    int threads = 1;   int stride = 32;
    int cacheLines = 1;
    intAddOne<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // Test.
  { int testNum = 5;    int blocks = 1;
    int threads = 32;   int stride = 32;
    int cacheLines = 32;
    intAddOne<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // Test
  { int testNum = 6;    int blocks = 1;
    int threads = 32;    int stride = 2;
    int cacheLines = 1;
    intAddOneHalf<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // Test
  { int testNum = 7;    int blocks = 1;
    int threads = 32;    int stride = 1;
    int cacheLines = 2;
    intAddOneOff<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // These tests care about number of warps that printed. Not cacheLines!
  // Test
  { int testNum = 8;    int blocks = 1;
    int threads = 33;    int stride = 1;
    int cacheLines = 1; int warps = 2;
    intAddOne<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines);
    printf(printStrWarp, testNum, warps); }


  // Test
  { int testNum = 9;    int blocks = 1;
    int threads = 64;    int stride = 1;
    int cacheLines = 1;  int warps = 2;
    intAddOne<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines);
    printf(printStrWarp, testNum, warps); }

  // Test
  { int testNum = 10;    int blocks = 1;
    int threads = 65;    int stride = 1;
    int cacheLines = 1;  int warps = 3;
    intAddOne<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines);
    printf(printStrWarp, testNum, warps); }

  // Test
  { int testNum = 11;    int blocks = 1;
    int threads = 33;    int stride = 1;
    int cacheLines = 1;  int warps = 2;
    intAddOneEvens<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines);
    printf(printStrWarp, testNum, warps); }

  // Test
  { int testNum = 12;    int blocks = 1;
    int threads = 33;    int stride = 1;
    int cacheLines = 1;  int warps = 1;
    intAddOneOdds<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines);
    printf(printStrWarp, testNum, warps); }

  // Check to see if different threads per warp can be the reduce thread.
  // Test
  { int testNum = 13;    int blocks = 1;
    int threads = 64;    int stride = 1;
    int cacheLines = 1;  int warps = 2;
    intAddOneDiff<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines);
    printf(printStrWarp, testNum, warps); }

  // Test
  { int testNum = 14;    int blocks = 3;
    int threads = 32;    int stride = 1;
    int cacheLines = 1;  int warps = 3;
    intAddOneSame<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines);
    printf(printStrWarp, testNum, warps); }

  // Test
  { int testNum = 15;    int blocks = 1;
    int threads = 96;    int stride = 1;
    int cacheLines = 1;  int warps = 3;
    intAddOneSame<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines);
    printf(printStrWarp, testNum, warps); }

  hipMemcpy(x, xDev, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipFree(x);
}

/**
 * Tests for accessing chars.
 */
void charTests(){
  char x[N] = {0};
  char* xDev;
  hipMalloc(&xDev, sizeof(char) * N);
  hipMemcpy(xDev, x, sizeof(char)* N, hipMemcpyHostToDevice);

  printf("[Char Tests]\n\n");
  // Run multiple tests to ensure our pass is working!
  // Synchonize necessary so that CPU waits for kernel to finish before printing.
  
  // Test. Needs 32 bytes of continuous memory.
  { int testNum = 1;    int blocks = 1;
    int threads = 32;   int stride = 1;
    int cacheLines = 1;
    charAddOne<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // Test. Needs 63 bytes of continuous memory.
  { int testNum = 2;    int blocks = 1;
    int threads = 32;   int stride = 2;
    int cacheLines = 1;
    charAddOne<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // Test. Needs 125 bytes of continuous memory.
  { int testNum = 3;    int blocks = 1;
    int threads = 32;   int stride = 4;
    int cacheLines = 1;
    charAddOne<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // Test. Needs 249 bytes of continuous memory.
  { int testNum = 4;    int blocks = 1;
    int threads = 32;   int stride = 8;
    int cacheLines = 2;
    charAddOne<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // Test. Needs 121 bytes of continuous memory.
  { int testNum = 5;    int blocks = 1;
    int threads = 16;   int stride = 8;
    int cacheLines = 1;
    charAddOne<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // Test. One access per thread.
  { int testNum = 6;    int blocks = 1;
    int threads = 32;   int stride = 128;
    int cacheLines = 32;
    charAddOne<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // Test. Needs 113 bytes of continuous memory.
  { int testNum = 7;    int blocks = 1;
    int threads = 8;    int stride = 16;
    int cacheLines = 1;
    charAddOne<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // Test. Needs 278 bytes of continuous memory.
  { int testNum = 8;    int blocks = 1;
    int threads = 32;   int stride = 9;
    int cacheLines = 3;
    charAddOne<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // Test. Needs 1 byte of continuous memory.
  { int testNum = 9;    int blocks = 5;
    int threads = 1;    int stride = 1;
    int cacheLines = 1;
    charAddOne<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // Test.
  { int testNum = 10;    int blocks = 5;
    int threads = 2;    int stride = 16;
    int cacheLines = 1;
    charAddOne<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // Test.
  { int testNum = 11;    int blocks = 5;
    int threads = 8;    int stride = 4;
    int cacheLines = 1;
    charAddOne<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // Test.
  { int testNum = 12;    int blocks = 1;
    int threads = 8;    int stride = 4;
    int cacheLines = 1;
    charAddOne<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // Test.
  { int testNum = 13;    int blocks = 1;
    int threads = 9;    int stride = 17;
    int cacheLines = 2;
    charAddOne<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }

  // Test.
  { int testNum = 14;    int blocks = 1;
    int threads = 8;    int stride = 50;
    int cacheLines = 3;
    charAddOne<<<blocks, threads >>>(xDev, stride);
    hipDeviceSynchronize(); printf(printStr, testNum, cacheLines); }


  hipMemcpy(x, xDev, sizeof(char) * N, hipMemcpyDeviceToHost);
  hipFree(x);
}

/**
 * Elements accessing continous memory from 1 byte data structure.
 * No uncoalesced accesses expected up to 128 byte ranges.
 * For 32 threads running that is 4 * threadIdx.x
 */
__global__ void charAddOne(char* array, int stride){
  int index = stride * threadIdx.x;
  array[index] = array[index] + 1;
}

/**
 * Elements accessing continous memory from 4 byte data structure.
 * No uncoalesced accesses expected up to 128 byte ranges.
 */
__global__ void intAddOne(int* array, int stride){
  int index = stride * threadIdx.x;
  array[index] = array[index] + 1;
}

/**
 * Elements accessing continous memory from 4 byte data structure.
 // Only evens!
 */
__global__ void intAddOneEvens(int* array, int stride){
  int index = stride * threadIdx.x;
  if(threadIdx.x % 2 == 0){
    array[index] = array[index] + 1;
  }
}

/**
 * Elements accessing continous memory from 4 byte data structure.
 // Only odds.
 */
__global__ void intAddOneOdds(int* array, int stride){
  int index = stride * threadIdx.x;
  if(threadIdx.x % 2 == 1){
    array[index] = array[index] + 1;
  }
}

/**
 * Elements accessing continous memory from 4 byte data structure.
 * Offset by two to test alignment.
 */
__global__ void intAddOneOff(int* array, int stride){
  int index = stride * threadIdx.x;
  // Pointer arithmetic...
  *(array + index - 1) = *(array + index - 1) + 1;
}

/**
 * Elements accessing continous memory from 4 byte data structure.
 * Only even threads running!
 */
__global__ void intAddOneHalf(int* array, int stride){
  int index = stride * threadIdx.x;
  if(index < 16)
    array[index] = array[index] + 1;
}

/**
 * Elements accessing continous memory from 4 byte data structure.
 * In here we have different elements per warp being the reduce thread.
 * on the first warp [0 - 31] the 0th thread is the reduce thread.
 * on the second warp [32 - 63] the 48th thread is the reduce thread.
 */
__global__ void intAddOneDiff(int* array, int stride){
  int index = stride * threadIdx.x;
  if(index < 16 || index >= 48)
    array[index] = array[index] + 1;
}

/**
 * Elements accessing continous memory from 4 byte data structure.
 * Same memory location through += operator.
 */
__global__ void intAddOneSame(int* array, int stride){
  int index = stride * threadIdx.x;
  array[index] += 1;
}

/**
 * Elements accessing continous memory from 8 byte data structure.
 */
__global__ void doubleAddOne(double* array, int stride){
  int index = stride * threadIdx.x;
  array[index] = array[index] + 1;
}


/**
 * Elements accessing continous memory from 12 byte data structure.
 */
__global__ void structAddOneX(struct myStruct* array, int stride){
  int index = stride * threadIdx.x;
  array[index].x = array[index].x + 1;
}

/**
 * Elements accessing continous memory from 12 byte data structure.
 */
__global__ void structAddOneY(struct myStruct* array, int stride){
  int index = stride * threadIdx.x;
  array[index].y = array[index].y + 1;
}

/**
 * Elements accessing continous memory from 12 byte data structure.
 */
__global__ void structAddOneZ(struct myStruct* array, int stride){
  int index = stride * threadIdx.x;
  array[index].z = array[index].z + 1;
}

