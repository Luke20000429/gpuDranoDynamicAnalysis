#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> 
#include <iostream>


#define checkCudaErrors(call)                                 \
  do {                                                        \
    hipError_t err = call;                                   \
    if (err != hipSuccess) {                                 \
      printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, \
             hipGetErrorString(err));                        \
      exit(EXIT_FAILURE);                                     \
    }                                                         \
  } while (0)


__global__ void axpy(float a, float* x, float* y) {
  y[threadIdx.x] = a * x[threadIdx.x];
}

int main(int argc, char* argv[]) {
  const int kDataLen = 4;

  float a = 2.0f;
  float host_x[kDataLen] = {1.0f, 2.0f, 3.0f, 4.0f};

  // Copy input data to device.
  float* x;
  float* y;
  checkCudaErrors(hipMallocManaged(&x, kDataLen * sizeof(float)));
  checkCudaErrors(hipMallocManaged(&y, kDataLen * sizeof(float)));
  for (int i = 0; i < kDataLen; ++i) {
    x[i] = host_x[i];
  }

  int device = -1;
  hipGetDevice(&device);
  hipMemPrefetchAsync(x, kDataLen * sizeof(float), device, NULL);
  hipMemPrefetchAsync(y, kDataLen * sizeof(float), device, NULL);

  // Launch the kernel.
  axpy<<<1, kDataLen>>>(a, x, y);

  // Copy output data to host.
  checkCudaErrors(hipDeviceSynchronize());

  // Print the results.
  for (int i = 0; i < kDataLen; ++i) {
    std::cout << "y[" << i << "] = " << y[i] << "\n";
  }

  checkCudaErrors(hipDeviceReset());
  return 0;
}

