
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

typedef uint64_t int64;
#define warpSizeD 32

/* =====================================================================================*/
/** Given an integer representing a 32-entry array of bits, return the nth bit of the
 * array as either 0 or 1, and a number of bits to shift.
 */
__device__ int getNthBit(unsigned int bitArray, int nth){
  return 1 & (bitArray >> nth);
}
/* =====================================================================================*/
/**
 * Return the lane id of the thread.
 */
static __device__ __inline__ uint32_t getLaneId(){
  uint32_t laneId;
  asm volatile("mov.u32 %0, %%laneid;" : "=r"(laneId));
  return (int)laneId;
}
/* =====================================================================================*/
/**
 * Function to count the number of unique cache lines needed for each load or store.
 * Prints information about location of load/store and number of cache lines needed.
 * @param: address of load/store we want to compute for.
 * @param: name of module where this load/store resides.
 * @param: function name where this load/store resides.
 * @param: either "load" or "store"
 * @param: a unique integer given to each invocation to this function to differentiate
 *         different dynamic instructions.
 */
__device__ void countCacheLines(void* addressP, char* moduleName, char* functionName,
                                char* loadOrStore, int lineNum, int columnNum,
                                int dynamicId, int typeSize){
  // Not a global memory address.
  if(1 != __isGlobal(addressP))
    return;

  /* Not all threads may be active in this function. We use the cuda ballot() function to
     figure out which treads are currently active. */
  int activeThreads =__ballot(1);
  // Our address will be used as a "None" value. Since we know it will not cause
  // problems when counting unique values. Notice this will only matter for the
  // reduce thread.
  int64 address = (int64) addressP;

  // Array to hold the addresses of all the threads. Twice as big as the warp
  // since we want the starting (min adress) and ending address (max adress) for every
  // read, that is, all the bytes a single thread is accessing.
  int64 addrArray[2 * warpSizeD];

  // Thread to gather values across threads.
  int reduceThread = -1;
  for(int i = 0; i < warpSizeD; i++)
    if(getNthBit(activeThreads, i) == 1){
      reduceThread = i;
      break;
    }

  // Shuffle values from all threads to our addrArray. Shuffling is undefined if we ask an
  // unactive thread. So we only query active threads.
  for(int i = 0; i < warpSizeD; i++){
    if(getNthBit(activeThreads, i) == 0)
      addrArray[2 * i] = address;
    else{
      // Break our shuffle into higher and lower order bits.
      int hob = (int)(address >> 32);
      int lob = 0xFFFFFFFF & address;
      hob = __shfl(hob, i);
      lob = __shfl(lob, i);
      addrArray[2 * i] = (((int64) hob) << 32) | (int64) lob;
    }
  }

  if(reduceThread == getLaneId()){
    // Number of unique cache lines.
    int count = 1;
    // Every other thread will represent the max address that
    // is accessed. We compute (address + typeSize - 1) for those.
    for(int i = 0; i < warpSizeD; i++)
      addrArray[2 * i + 1] = addrArray[2 * i] + typeSize - 1;

    int64 min = addrArray[0];

    // Ignore alignment issues by finding minimum element and subtracting from all others.
    for(int i = 0; i < 2 * warpSizeD; i++)
      if(min > addrArray[i])
        min = addrArray[i];
    for(int i = 0; i < 2 * warpSizeD; i++)
      addrArray[i] = (addrArray[i] - min) >> 7;

    // We access this value instead of address as it has been divided by 128.
    int64 myNone = addrArray[0];
    // Count unique elements.
    for(int i = 0; i < 2 * warpSizeD; i++)
      if(addrArray[i] != myNone){       // Skip inactive threads.
        int64 current = addrArray[i];
        count++;
        // Iterate through rest of addrArray "none-ing out" entries that match current.
        for(int j = i + 1; j < 2 * warpSizeD; j++)
          if(addrArray[j] == current)
            addrArray[j] = myNone;
      }

    // 'DA__' is needed so we have a unique identifier to grep from the program's
    // output.
    char* str = "DA__\t%s\t%s\t%d\t%s\t%d\t%d\t%d\n";
    printf(str, moduleName, functionName, dynamicId, loadOrStore,
           lineNum, columnNum, count);
  }

  return;
}
/* =====================================================================================*/