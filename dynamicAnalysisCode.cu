
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

typedef uint64_t int64;

/* =====================================================================================*/
/** Given an integer representing a 32-entry array of bits, return the nth bit of the
 * array as either 0 or 1, and a number of bits to shift.
 */
__device__ int getNthBit(unsigned int bitArray, int nth){
  return 1 & (bitArray >> nth);
}
/* =====================================================================================*/
/**
 * Function to count the number of unique cache lines needed for each load or store.
 * Prints information about location of load/store and number of cache lines needed.
 * @param: address of load/store we want to compute for.
 * @param: name of module where this load/store resides.
 * @param: function name where this load/store resides.
 * @param: either "load" or "store"
 * @param: a unique integer given to each invocation to this function to differentiate
 *         different dynamic instructions.
 */
__device__ void countCacheLines(void* addressP, char* moduleName, char* functionName,
                                char* loadOrStore, int lineNum, int columnNum,
                                int dynamicId, int typeSize){
  // Not a global memory address.
  if(1 != __isGlobal(addressP))
    return;

  /* Not all threads may be active in this function. We use the cuda ballot() function to
     figure out which treads are currently active. */
  int activeThreads =__ballot(1);
  // Our address will be used as a "None" value. Since we know it will not cause
  // problems when counting unique values. Notice this will only matter for the
  // reduce thread.
  int64 address = (int64) addressP;
  //  printf("Hello from Thread: %d\n", threadIdx.x);

  // Array to hold the addresses of all the threads. We make it one bigger to hold
  // (max + typeSize - 1). This is to account for the size of our data when checking
  // for uncoalesced accesses.
  int64 addrArray[33];

  // Thread to gather values across threads.
  int reduceThread = -1;
  for(int i = 0; i < 32; i++)
    if(getNthBit(activeThreads, i) == 1){
      reduceThread = i;
      break;
    }

  // Shuffle values from all threads to our addrArray. Shuffling is undefined if we ask an
  // unactive thread. So we only query active threads.
  for(int i = 0; i < 32; i++){
    if(getNthBit(activeThreads, i) == 0)
      addrArray[i] = address;
    else{
      // Break our shuffle into higher and lower order bits.
      int hob = (int)(address >> 32);
      int lob = 0xFFFFFFFF & address;
      hob = __shfl(hob, i);
      lob = __shfl(lob, i);
      addrArray[i] = (((int64) hob) << 32) | (int64) lob;
    }
  }

  // We are computing based on warps, but thread id's go past 32. So we must modulo 
  // around.
  if(reduceThread == (threadIdx.x % 32)){
    // Number of unique cache lines.
    int count = 1;
    int64 myNone = address >> 7;

    // We must account for the size of the data that we are accessing. We find the
    // maximum element and add typeSize - 1 to it.
    int64 max = address;
    /* By definition, the reduceThread is the first active thread. Start after him. */
    for(int i = reduceThread + 1; i < 32; i++)
      if(max < addrArray[i])
        max = addrArray[i];
    addrArray[32] = (max + typeSize - 1);

    // Divide all threads by 128
    for(int i = reduceThread + 1; i < 33; i++){
      addrArray[i] >>= 7;
    }
    // Count unique elements.
    for(int i = reduceThread + 1; i < 33; i++)
      if(addrArray[i] != myNone){       // Skip inactive threads.
        int64 current = addrArray[i];
        count++;
        // Iterate through rest of addrArray "none-ing out" entries that match current.
        for(int j = i + 1; j < 33; j++)
          if(addrArray[j] == current)
            addrArray[j] = myNone;
      }

    // 'DA__' is needed so we have a unique identifier to grep from the program's
    // output.
    char* str = "DA__\t%s\t%s\t%d\t%s\t%d\t%d\t%d\n";
    printf(str, moduleName, functionName, dynamicId, loadOrStore,
           lineNum, columnNum, count);
  }

  return;
}
/* =====================================================================================*/