
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

typedef int64_t int64;

/* =====================================================================================*/
/** Given an integer representing a 32-entry array of bits, return the nth bit of the         
 * array as either 0 or 1, and a number of bits to shift.                                     
 */
__device__ int getNthBit(unsigned int bitArray, int nth){
  return 1 & (bitArray >> nth);
}
/* =====================================================================================*/
/**
 * Function to count the number of unique cache lines needed for each load or store.
 * Prints information about location of load/store and number of cache lines needed.
 * @param: address of load/store we want to compute for.
 * @param: name of module where this load/store resides.
 * @param: function name where this load/store resides.
 * @param: either "load" or "store"
 */
__device__ void countCacheLines(void* address, char* moduleName, char* functionName,
                                char* loadOrStore, int lineNum, int columnNum){
  // Not a global memory address.
  if(1 != __isGlobal(address))
    return;

  /* Not all threads may be active in this function. We use the cuda ballot() function to
     figure out which treads are currently active. */
  int activeThreads =__ballot(1);
  // Divide by 128 to find unique adresses.
  int64 div = ((int64) address) >> 7;
  // Thread to gather values across threads.
  int reduceThread = -1;
  for(int i = 0; i < 32; i++){
    if(getNthBit(activeThreads, i) == 1){
      reduceThread = i;
      break;
    }
  }

  // Array to hold the value of all the threads.
  int64 array[32];

  /*Shuffle values from all threads to our array. Shuffling is undefined if we ask an
    unactive thread. So we only query active threads. */
  for(int i = 0; i < 32; i++){
    // Our div value will be used as a "None" value. Since we know it will not cause
    // problems when counting unique values.
    if(getNthBit(activeThreads, i) == 0)
      array[i] = div;
    else{
      // Break our shuffle into higher and lower order bits.
      int hob = (int)((div >> 32));
      int lob = 0xFFFFFFFF & div;
      hob = __shfl(hob, i);
      lob = __shfl(lob, i);
      array[i] = (((int64) hob) << 32) | (int64) lob;
    }
  }

  // Find unique and print if you're the reduceThread.
  if(reduceThread == threadIdx.x){
    int count = 1;
    // This is also our value at array[reduceThread].
    int myNone = div;
    /* By definition, the reduceThread is the first active thread. Start after him. */
    for(int i = reduceThread + 1; i < 32; i++){
      // Skip inactive threads.
      if(array[i] != myNone){
        int current = array[i];
        count++;
        // Iterate through rest of array "non-ing out" entries that match current.
        for(int j = i + 1; j < 32; j++)
          if(array[j] == current)
            array[j] = myNone;
      }
    }


    // ModuleName FunctionName LineNumber columnNumber UniqueCount
    char* str = "%s\t%s\t%s\t%d\t%d\t%d\n";
    printf(str, moduleName, functionName, loadOrStore, lineNum, columnNum, count);
  }

  return;
}
/* =====================================================================================*/